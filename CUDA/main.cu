// Copyright (c) 2016 Nicolas Weber and Sandra C. Amend / GCC / TU-Darmstadt. All rights reserved. 
// Use of this source code is governed by the BSD 3-Clause license that can be
// found in the LICENSE file.
#define _USE_MATH_DEFINES 
#include <math.h>
#include <iostream>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>

#define THREADS 128
#define WSIZE 32
#define TSIZE (THREADS / WSIZE)

#define TX threadIdx.x
#define PX (blockIdx.x * TSIZE + (TX / WSIZE))
#define PY blockIdx.y

#define WTHREAD	(TX % WSIZE)
#define WARP	(TX / WSIZE)

//-------------------------------------------------------------------
// SHARED
//-------------------------------------------------------------------
struct Params {
	uint32_t oWidth;
	uint32_t oHeight;
	uint32_t iWidth;
	uint32_t iHeight;
	float pWidth;
	float pHeight;
	float lambda;
};

//-------------------------------------------------------------------
// DEVICE
//-------------------------------------------------------------------
__device__ __forceinline__ void normalize(float4& var) {
	var.x /= var.w;
	var.y /= var.w;
	var.z /= var.w;
	var.w = 1.0f;
}

//-------------------------------------------------------------------
__device__ __forceinline__ void add(float4& output, const uchar3& color, const float factor) {
	output.x += color.x * factor;	
	output.y += color.y * factor;	
	output.z += color.z * factor;	
	output.w += factor;
}

//-------------------------------------------------------------------
__device__ __forceinline__ void add(float4& output, const float4& color) {
	output.x += color.x;
	output.y += color.y;
	output.z += color.z;
	output.w += color.w;
}

//-------------------------------------------------------------------
__device__ __forceinline__ float lambda(const Params p, const float dist) {
	if(p.lambda == 0.0f)
		return 1.0f;
	else if(p.lambda == 1.0f)
		return dist;

	return pow(dist, p.lambda);
}

//-------------------------------------------------------------------
__device__ __forceinline__ void operator+=(float4& output, const float4 value) {
	output.x += value.x;
	output.y += value.y;
	output.z += value.z;
	output.w += value.w;
}

//-------------------------------------------------------------------
struct Local {
	float sx, ex, sy, ey;
	uint32_t sxr, syr, exr, eyr, xCount, yCount, pixelCount;

	__device__ __forceinline__ Local(const Params& p) {
		sx			= fmaxf(PX		* p.pWidth, 0.0f);
		ex			= fminf((PX+1)	* p.pWidth, p.iWidth);
		sy			= fmaxf(PY		* p.pHeight, 0.0f);
		ey			= fminf((PY+1)	* p.pHeight, p.iHeight);

		sxr			= (uint32_t)floor(sx);
		syr			= (uint32_t)floor(sy);
		exr			= (uint32_t)ceil(ex);
		eyr			= (uint32_t)ceil(ey);
		xCount		= exr - sxr;
		yCount		= eyr - syr;
		pixelCount	= xCount * yCount;
	}
};

//-------------------------------------------------------------------
__device__ __forceinline__ float contribution(const Local& l, float f, const uint32_t x, const uint32_t y) {
	if(x < l.sx)		f *= 1.0f - (l.sx - x);
	if((x+1.0f) > l.ex)	f *= 1.0f - ((x+1.0f) - l.ex);
	if(y < l.sy)		f *= 1.0f - (l.sy - y);
	if((y+1.0f) > l.ey)	f *= 1.0f - ((y+1.0f) - l.ey);
	return f;
}

//-------------------------------------------------------------------
// taken from: https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/
__device__ __forceinline__ float4 __shfl_down(const float4 var, const uint32_t srcLane, const uint32_t width = 32) {
	float4 output;
	output.x = __shfl_down(var.x, srcLane, width);
	output.y = __shfl_down(var.y, srcLane, width);
	output.z = __shfl_down(var.z, srcLane, width);
	output.w = __shfl_down(var.w, srcLane, width);
	return output;
}

//-------------------------------------------------------------------
__device__ __forceinline__ void reduce(float4& value) {
	value += __shfl_down(value, 16);
	value += __shfl_down(value, 8);
	value += __shfl_down(value, 4);
	value += __shfl_down(value, 2);
	value += __shfl_down(value, 1);
}

//-------------------------------------------------------------------
__device__ __forceinline__ float distance(const float4& avg, const uchar3& color) {
	const float x = avg.x - color.x;
	const float y = avg.y - color.y;
	const float z = avg.z - color.z;

	return sqrt(x * x + y * y + z * z) / 441.6729559f; // L2-Norm / sqrt(255^2 * 3)
}

//-------------------------------------------------------------------
__global__ void kernelGuidance(const uchar3* __restrict__ input, uchar3* __restrict__ patches, const Params p) {
    if(PX >= p.oWidth || PY >= p.oHeight) return;

	// init
	const Local l(p);
	float4 color = {0};

	// iterate pixels
	for(uint32_t i = WTHREAD; i < l.pixelCount; i += WSIZE) {
		const uint32_t x = l.sxr + (i % l.xCount);
		const uint32_t y = l.syr + (i / l.xCount);
		 
		float f = contribution(l, 1.0f, x, y);	

		const uchar3& pixel = input[x + y * p.iWidth];
		add(color, make_float4(pixel.x * f, pixel.y * f, pixel.z * f, f));
	}

	// reduce warps
	reduce(color);

	// store results
	if((TX % 32) == 0) {
		normalize(color);
		patches[PX + PY * p.oWidth] = make_uchar3(color.x, color.y, color.z);
	}
}

//-------------------------------------------------------------------
__device__ __forceinline__ float4 calcAverage(const Params& p, const uchar3* __restrict__ patches) {
	const float corner	= 1.0;
	const float edge	= 2.0;
	const float center	= 4.0;

	// calculate average color
	float4 avg = {0};

	// TOP
	if(PY > 0) {
		if(PX > 0) 
			add(avg, patches[(PX - 1) + (PY - 1) * p.oWidth], corner);

		add(avg, patches[(PX) + (PY - 1) * p.oWidth], edge);
	
		if((PX+1) < p.oWidth)
			add(avg, patches[(PX + 1) + (PY - 1) * p.oWidth], corner);
	}

	// LEFT
	if(PX > 0) 
		add(avg, patches[(PX - 1) + (PY) * p.oWidth], edge);

	// CENTER
	add(avg, patches[(PX) + (PY) * p.oWidth], center);
	
	// RIGHT
	if((PX+1) < p.oWidth)
		add(avg, patches[(PX + 1) + (PY) * p.oWidth], edge);

	// BOTTOM
	if((PY+1) < p.oHeight) {
		if(PX > 0) 
			add(avg, patches[(PX - 1) + (PY + 1) * p.oWidth], corner);

		add(avg, patches[(PX) + (PY + 1) * p.oWidth], edge);
	
		if((PX+1) < p.oWidth)
			add(avg, patches[(PX + 1) + (PY + 1) * p.oWidth], corner);
	}

	normalize(avg);

	return avg;
}

//-------------------------------------------------------------------
__global__ void kernelDownsampling(const uchar3* __restrict__ input, const uchar3* __restrict__ patches, const Params p, uchar3* __restrict__ output) {
    if(PX >= p.oWidth || PY >= p.oHeight) return;

	// init
	const Local l(p);
	const float4 avg = calcAverage(p, patches);

	float4 color = {0};

	// iterate pixels
	for(uint32_t i = WTHREAD; i < l.pixelCount; i += WSIZE) {
		const uint32_t x = l.sxr + (i % l.xCount);
		const uint32_t y = l.syr + (i / l.xCount);

		const uchar3& pixel = input[x + y * p.iWidth];
		float f = distance(avg, pixel);
		
		f = lambda(p, f);
		f = contribution(l, f, x, y);

		add(color, pixel, f);
	}

	// reduce warp
	reduce(color);

	if(WTHREAD == 0) {
		uchar3& ref = output[PX + PY * p.oWidth];

		if(color.w == 0.0f)
			ref = make_uchar3((unsigned char)avg.x, (unsigned char)avg.y, (unsigned char)avg.z);
		else {
			normalize(color);
			ref = make_uchar3((unsigned char)color.x, (unsigned char)color.y, (unsigned char)color.z);
		}
	}
}

//-------------------------------------------------------------------
// HOST
//-------------------------------------------------------------------
void check(hipError_t err) {
	if(err != hipSuccess) {
		std::cerr << "CUDA_ERROR: " << (int)err << " " << hipGetErrorName(err) << ": " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
}

//-------------------------------------------------------------------
void run(const Params& i, const void* hInput, void* hOutput) {
	// calc sizes
	const size_t sInput		= sizeof(uchar3) * i.iWidth * i.iHeight;
	const size_t sOutput	= sizeof(uchar3) * i.oWidth * i.oHeight;
	const size_t sGuidance	= sizeof(uchar3) * i.oWidth * i.oHeight;

	// alloc GPU
	uchar3* dInput = 0, *dOutput = 0, *dGuidance = 0;
	
	check(hipMalloc(&dInput, sInput));
	check(hipMalloc(&dOutput, sOutput));
	check(hipMalloc(&dGuidance, sGuidance));

	// copy data
	check(hipMemcpy(dInput, hInput, sInput, hipMemcpyHostToDevice));

	// launch config
	const dim3 threads(THREADS, 1, 1); // 4 warps, 1 warp per patch
	const dim3 blocks((uint32_t)std::ceil(i.oWidth / (double)TSIZE), i.oHeight, 1);
	
	// execute kernels
	kernelGuidance		<<<blocks, threads>>>(dInput, dGuidance, i);
	kernelDownsampling	<<<blocks, threads>>>(dInput, dGuidance, i, dOutput);

	// copy data
	check(hipMemcpy(hOutput, dOutput, sOutput, hipMemcpyDeviceToHost));

	// free GPU
	check(hipFree(dInput));
	check(hipFree(dOutput));
	check(hipFree(dGuidance));
	
	// reset device
	//check(cudaDeviceReset());
}